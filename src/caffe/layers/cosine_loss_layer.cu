#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Cos(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = cos(in[index]);
  }
}

template <typename Dtype>
__global__ void Sin(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = sin(in[index]);
  }
}

template <typename Dtype>
void CosineLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Cos<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, diff_.gpu_data(), tmp_.mutable_gpu_data());
  Dtype sum; caffe_gpu_dot(count, tmp_.gpu_data(), ones_.gpu_data(), &sum);
  Dtype loss = (bottom[0]->num()*bottom[0]->channels() - sum) / bottom[0]->num() / Dtype(2);
  (*top)[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void CosineLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / (*bottom)[i]->num() / Dtype(2);
      Sin<Dtype><<<CAFFE_GET_BLOCKS((*bottom)[i]->count()), CAFFE_CUDA_NUM_THREADS>>>(
            (*bottom)[i]->count(), diff_.gpu_data(), tmp_.mutable_gpu_data());
      caffe_gpu_axpby(
          (*bottom)[i]->count(),              // count
          alpha,                              // alpha
          tmp_.gpu_data(),                    // a
          Dtype(0),                           // beta
          (*bottom)[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_CLASS(CosineLossLayer);

}  // namespace caffe
